#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>              // core runtime API
#include <cuda_surface_types.h>        // surface object type
#include <surface_indirect_functions.h>   // <-- declares surf2DLayeredRead/Write
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <optix.h>
#include "gdt/math/vec.h"
#include "LaunchParams.h"

using namespace gdt;
using namespace osc;

extern "C" __constant__ LaunchParams lp;   // declared once in host code
/*---------------------------------------------------------------------*/
/*  packing helper                                                     */
/*---------------------------------------------------------------------*/
__device__ __forceinline__ unsigned int packR11G11B10(vec3f c)
{
    c = max(c, vec3f(0.f));
    uint16_t rx = __half_as_ushort(__float2half_rn(c.x));
    uint16_t gy = __half_as_ushort(__float2half_rn(c.y));
    uint16_t bz = __half_as_ushort(__float2half_rn(c.z));
    return ((rx >> 5) & 0x07FF) | (((gy >> 5) & 0x07FF) << 11)
        | (((bz >> 6) & 0x03FF) << 22);
}
/*---------------------------------------------------------------------*/
/*  ray-generation : one thread = one destination texel                */
/*---------------------------------------------------------------------*/
extern "C" __global__
void __raygen__pack_texels()
{
    const unsigned flat = optixGetLaunchIndex().x;
    // only run exactly lp.texels threads
    if (flat >= lp.texels) return;

    const unsigned faceRes = 6u;
    const unsigned facesPerProbe = 6u;
    const unsigned probes = lp.probeCount;
    const unsigned tilesPerFace = faceRes * faceRes;    // 36
    const unsigned arraySize = probes * facesPerProbe;

    // decompose flat  probe, face, u, v
    const unsigned probe = flat / (facesPerProbe * tilesPerFace);
    const unsigned face = (flat / tilesPerFace) % facesPerProbe;
    const unsigned texel = flat % tilesPerFace;
    const unsigned u = texel % faceRes;
    const unsigned v = texel / faceRes;

    const unsigned layer = probe * facesPerProbe + face;
    if (probe < probes && face < facesPerProbe && layer < arraySize)
    {
        // compute the index into your linear lp.irrAccum array:
        const unsigned int idx = layer * tilesPerFace + v * faceRes + u;

        // fetch and pack
        vec4f f = lp.irrAccum[idx];
        uint32_t packed = packR11G11B10(vec3f(f.x,f.y,f.z));

        // write into the layered surface
        surf2DLayeredwrite<uint32_t>(
            packed,
            lp.irrSurf,
            int(u), int(v),
            int(layer)
        );


    }

    
}