#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_surface_types.h>
#include <hip/hip_fp16.h>
#include <optix_device.h>
#include "gdt/math/vec.h"
#include "LaunchParams.h"
#include <cstdint>
using namespace gdt;
using namespace osc;

extern "C" __constant__ LaunchParams lp;

__device__ __forceinline__ vec3f fibonacciDir(uint32_t i)
{
    const float k = float(i) + 0.5f, n = 256.f, z = 1.f - 2.f * k / n;
    const float r = sqrtf(max(0.f, 1.f - z * z)), phi = k * 2.39996323f;
    return vec3f(cosf(phi) * r, sinf(phi) * r, z);
}

struct FaceUV { unsigned int face, u, v; };
__device__ __forceinline__ FaceUV dirToCube(vec3f d)
{
    vec3f a = abs(d); unsigned int f; float sc, tc, ma;
    if (a.x >= a.y && a.x >= a.z) { f = d.x > 0 ? 0 : 1; ma = a.x; sc = d.z; tc = d.y * (d.x > 0 ? 1 : -1); }
    else if (a.y >= a.x && a.y >= a.z) { f = d.y > 0 ? 2 : 3; ma = a.y; sc = d.x; tc = d.z * (d.y > 0 ? -1 : 1); }
    else { f = d.z > 0 ? 4 : 5; ma = a.z; sc = d.z > 0 ? -d.x : d.x; tc = d.y; }
    unsigned int u = min(unsigned int((sc / ma * 0.5f + 0.5f) * 6.f), 5u);
    unsigned int v = min(unsigned int((tc / ma * 0.5f + 0.5f) * 6.f), 5u);
    return { f,u,v };
}

/* R11G11B10 packing  (returns 32-bit uint) */
__device__ __forceinline__ unsigned int packR11G11B10(vec3f c)
{
    uint16_t rx = __half_as_ushort(__float2half_rn(c.x)); // 5 exponent, 6 mant.
    uint16_t gy = __half_as_ushort(__float2half_rn(c.y));
    uint16_t bz = __half_as_ushort(__float2half_rn(c.z)); // only 5 mantissa bits
    return ((rx >> 5) & 0x7FF) | (((gy >> 5) & 0x7FF) << 11)
        | (((bz >> 6) & 0x3FF) << 22);
}

/*???????????????? payload helpers ?????????*/
struct PayloadRGB { vec3f L; };
static __device__ __forceinline__ void setPayload(const vec3f& L) {
    optixSetPayload_0(__float_as_uint(L.x));
    optixSetPayload_1(__float_as_uint(L.y));
    optixSetPayload_2(__float_as_uint(L.z));
}

/*???????????????? miss � simple sky ???????*/
extern "C" __global__ void __miss__radiance()
{
    setPayload(vec3f(0.03f, 0.04f, 0.05f));
}

/*???????????????? closest-hit radiance ????*/
extern "C" __global__ void __closesthit__radiance()
{
    /* your real shading here � for now constant white */
    setPayload(vec3f(1.f));
}

/*???????????????? ray-gen 0 : accumulate ??*/
extern "C" __global__
void __raygen__ddgi_accum()
{
    const unsigned int rayID = optixGetLaunchIndex().x;   // 0..255
    const unsigned int probeID = optixGetLaunchIndex().y;

    unsigned int j = (rayID + 47u * lp.frameID) & 255u;
    vec3f dir = fibonacciDir(j);
    vec3f org = lp.probePos[probeID];

    /* payload registers */
    unsigned int p0 = 0, p1 = 0, p2 = 0;
    optixTrace(lp.tlas, org, dir,
        1e-3f, 1e20f, 0.0f,
        0xFF, OPTIX_RAY_FLAG_NONE,
        0, 1, 0,
        p0, p1, p2);

    vec3f L = { __uint_as_float(p0), __uint_as_float(p1), __uint_as_float(p2) };

    float  w = fabsf(dir.z) * (1.f / M_PI) / 256.f;
    vec3f  dE = L * w;

    FaceUV f = dirToCube(dir);
    unsigned int   idx = (probeID * 6u + f.face) * 36u + f.v * 6u + f.u;

    atomicAdd(&lp.irrAccum[idx].x, dE.x);
    atomicAdd(&lp.irrAccum[idx].y, dE.y);
    atomicAdd(&lp.irrAccum[idx].z, dE.z);
    atomicAdd(&lp.irrAccum[idx].w, 1.0f);
}